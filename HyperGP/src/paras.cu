#include "paras.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>


namespace py = pybind11;

void initialize_global_streams() {
    std::cout << "initialize_global_streams " << std::endl;
    for(int k = 0; k < STREAM_NUM_NDARRAY; ++k){
        hipStreamCreate(&streams[0][k]);
    }
}

// bool initialized = []() {
//     initialize_global_streams();
//     return true;
// }();

GlobalStreams gstreams;

PYBIND11_MODULE(global_streams, m){
    py::class_<GlobalStreams>(m, std::string("GlobalStreams").c_str());
    initialize_global_streams();
    gstreams.init(streams);
    m.def("get_gstreams", [](){ 
        return gstreams;
    });
}
