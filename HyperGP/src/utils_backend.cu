#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <pybind11/numpy.h>
#include <iostream>
#include <vector>
#include <unordered_map>

#include <thread>
#include <mutex>
#include <unistd.h>
#include <numeric>
#include <list>
#include <thrust/device_vector.h>

#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"

namespace py = pybind11;

typedef std::tuple<py::array_t<int>, std::vector<int>, std::vector<int>, std::vector<std::string>, std::vector<float>, int> transformer_ret;

namespace pygp_utils{

    #define PYBIND11_NO_ASSERT_GIL_HELD_INCREF_DECREF
    void getChilds(std::vector<int> const& node_arity, std::vector<std::vector<int>>& node_childs){
        
        std::vector<std::vector<int>> cur_arity_tmp(node_arity.size() * 2);
        cur_arity_tmp.push_back({0, node_arity[0]});
        
        int node_arity_ksize = node_arity.size();
        for(int i = 1; i < node_arity_ksize; ++i){
            int idx = cur_arity_tmp.back()[0];
            cur_arity_tmp.back()[1] -= 1;
            node_childs[idx].push_back(i);
            if (cur_arity_tmp.back()[1] == 0){
                cur_arity_tmp.pop_back();
            }
            if(node_arity[i] > 0){
                cur_arity_tmp.push_back({i, node_arity[i]});
            }
        }
    }

    #define PYBIND11_NO_ASSERT_GIL_HELD_INCREF_DECREF
    void getChilds(float* idxs_ptr, std::vector<int>& f_arity, int& idxs_size, std::vector<std::vector<int>>& node_childs, int& func_len){
        
        std::vector<std::vector<float>> cur_arity_tmp;
        cur_arity_tmp.reserve(idxs_size * 2);
        cur_arity_tmp.push_back({0, idxs_ptr[1]});
        register int cur_arity;
        for(int i = 1; i < idxs_size; ++i){
            if(cur_arity_tmp.size() > 0){
                cur_arity = cur_arity_tmp.back()[1];
                if(cur_arity > 0){
                    node_childs[cur_arity_tmp.back()[0]].push_back(i);
                    cur_arity -= 1;
                }
                if (cur_arity <= 0){
                    cur_arity_tmp.pop_back();
                }
                else{
                    cur_arity_tmp.back()[1] = cur_arity;
                }
            }
            if(idxs_ptr[i * 3 + 2] == 0){
                cur_arity_tmp.push_back({float(i), idxs_ptr[i * 3 + 1]});
            }
        }
    }
    
    std::mutex mtx, mtx_constval;

    template<typename scalar_t>
    void transformer(const std::tuple<std::vector<std::string>, std::vector<int>, int>* f_attrs, 
                     const std::vector<size_t>* ind_after_cashes, const std::vector<size_t>* idxs, size_t sym_set_ptr,
                     std::vector<std::vector<int>>* cash_list, const std::vector<std::vector<int>>& records, std::vector<float>* constants,
                     const std::vector<int>& paras, int* id_allocator, const std::tuple<int, int, int>& basic_info, std::vector<std::vector<std::vector<int>>>* exp_set,
                     std::vector<int>* record_posi, std::vector<std::string>* record_strs, int* const_idx){
        int cur_ind = std::get<0>(basic_info);
        int cur_posi = std::get<1>(basic_info);
        int compute_unit = std::get<2>(basic_info);
        int arguments_num = paras[0], exec_len_max = paras[2], pset_funcs_num = paras[3];
        std::unordered_map<std::string, std::array<int, 2>> output;
        int node_size = 0;
        for(int i = cur_ind; i < compute_unit + cur_ind; ++i){
            node_size += (*idxs)[i * 2 + 1];
        }

        (*exp_set).reserve(node_size * exec_len_max);
        register std::string* sym_set;
        register bool pre_symset = false;
        if (sym_set_ptr == 0){
            sym_set = new std::string[node_size];
        }
        else{
            sym_set = (std::string*)sym_set_ptr + cur_posi;
            pre_symset = true;
        }
        
        register size_t idxs_size = compute_unit + cur_ind;
        
        std::vector<std::string> f_name = std::get<0>((*f_attrs));
        std::vector<int> f_arity;// = std::get<1>((*f_attrs));
        int func_len = std::get<2>((*f_attrs));
        int cur_expset_size = 0;
        int init_origin_posi = cur_posi;
        register int max_layer, child_size, idx, layer;
        std::string sym;
        std::vector<int> exps(exec_len_max);
        size_t ind_cashes_size;
        std::string sym_child;
        for(int k = cur_ind; k < idxs_size; ++k){
            float* idxs_ptr = (float*)((*idxs)[k * 2]);
            int idxs_ksize = (*idxs)[k * 2 + 1];
            std::vector<std::vector<int>> node_childs(idxs_ksize);
            
            if((*cash_list).size() > 0){
                int cash_list_ksize = (*cash_list)[k].size();
                std::vector<int>& cash_list_k = (*cash_list)[k];
                for(int i = 0; i < cash_list_ksize; ++i){
                    mtx.lock();
                    output[sym_set[cash_list_k[i]]] = {(*id_allocator), 0};
                    (*id_allocator) += 1;
                    mtx.unlock();
                }
            }
            getChilds(idxs_ptr, f_arity, idxs_ksize, node_childs, func_len);
            
            ind_cashes_size = (*ind_after_cashes)[k * 2 + 1];
            // if(ind_cashes_size == 1 && ind_after_cashes[k][0] != 0){
            //     ind_cashes_size = ind_after_cashes[k][0];
            // }
            int* preorder_offset = (int*)((*ind_after_cashes)[k * 2]);
            for(int ii = ind_cashes_size - 1; ii>=0; --ii){
                int i = preorder_offset[ii], iter_i = i + init_origin_posi;
                idx = int(idxs_ptr[i * 3]);
                child_size = node_childs[i].size();
                if(idxs_ptr[i * 3 + 2] == 0){
                    // assert (child_size == idxs_ptr[i * 3 + 1]);
                    // sym.reserve(100);
                    if(pre_symset){
                        sym = sym_set[i];
                    }
                    else{
                        sym = f_name[idx] + '(';
                    }

                    max_layer = 0;
                    
                    exps[0] = idx;
                    exps[1] = child_size;
                    for (int j = 0; j < child_size; ++j){
                        sym_child = sym_set[node_childs[i][j]];
                        layer = output[sym_child][1];
                        exps[j + 2] = output[sym_child][0];
                        
                        if(!pre_symset){
                            sym += sym_child + ", ";
                        }
                        if (layer > max_layer){
                            max_layer = layer;
                        }
                    }
                    if (!pre_symset){
                        sym = sym.erase(sym.size() - 2, 2) + ')';
                        sym_set[i] = sym;
                    }

                    if(output.count(sym) == 0 || i == 0){
                        
                        if (i == 0){
                            exps[child_size + 2] = arguments_num + k;
                            output[sym] = {arguments_num + k, max_layer + 1};
                        }
                        else{
                            /// [ ] TODO: record_dict should be replaced by list struct
                            mtx.lock();
                            exps[child_size + 2] = (*id_allocator);
                            output[sym] = {(*id_allocator), max_layer + 1};
                            (*id_allocator) += 1;
                            mtx.unlock();
                        }
                        
                        if (max_layer >= cur_expset_size){
                            (*exp_set).push_back({exps});
                            cur_expset_size += 1;
                        }
                        else{
                            (*exp_set)[max_layer].push_back(exps);
                        }
                    }

                }
                else{
                    /// [ ] TODO: unable to handle the self-define function.
                    
                    std::string node_str;
                    if(idxs_ptr[i * 3 + 2] == 1){
                        node_str = f_name[idx + pset_funcs_num];
                    }
                    else{
                        node_str = std::to_string(idxs_ptr[i * 3]);
                    }
                    if(output.count(node_str) == 0){
                        if(idxs_ptr[i * 3 + 2] == 1){
                            output[node_str] = {idx, 0};
                        }
                        else{
                            (*constants).push_back(idxs_ptr[i * 3]);
                            mtx_constval.lock();
                            output[node_str] = {-*const_idx - 1, 0};
                            *const_idx += 1;
                            mtx_constval.unlock();
                        }
                    }
                    if(!pre_symset){
                        sym_set[i] = node_str;
                    }
                }
            }
            // printf("ind_cashes_size: %d\n", ind_cashes_size);
            if(ind_cashes_size == 1){
                max_layer = 0;
                idx = idxs_ptr[0];
                exps[0] = pset_funcs_num - 1;
                exps[1] = 1;
                if(idxs_ptr[2] == 1){
                    exps[2] = idx;
                }
                else{
                    (*constants).push_back(idxs_ptr[0]);
                    mtx_constval.lock();
                    exps[2] = -*const_idx - 1;
                    *const_idx += 1;
                    mtx_constval.unlock();
                }
                exps[3] = arguments_num + k;
                
                if (0 >= cur_expset_size){
                    (*exp_set).push_back({exps});
                    cur_expset_size += 1;
                }
                else{
                    (*exp_set)[0].push_back(exps);
                }
            }
            int records_k_size = records[k].size();
            for(int i = 0; i < records_k_size; ++i){
                    
                std::string sym = sym_set[records[k][i]];
                (*record_posi).push_back(output[sym][0]);
                (*record_strs).push_back(sym);
                // printf("Here....%d, %d, %d, %d, %s\n", records[k][i], output[sym][0], cur_ind, compute_unit + cur_ind, sym.c_str());
            }
            init_origin_posi += idxs_ksize;
        }
        
        if(!pre_symset){
            delete[] sym_set;
        }
        // return transformer_ret(record_posi, record_strs, id_allocator);
    }

    void exec_sum(int* exec_len, std::vector<std::vector<std::vector<int>>>* exp_set){
        int exp_size1 = (*exp_set).size();
        for(int i = 0; i < exp_size1; ++i){
            *exec_len += (*exp_set)[i].size();
        }
    }
    void exec_cpy(size_t buf_ptr, std::vector<std::vector<int>>* exp_set){
        int* exp_final_set = (int*)buf_ptr;
        int exp_size2 = (*exp_set).size();
        for(int j = 0; j < exp_size2; ++j){
            int exp_size3 = (*exp_set)[j].size();
            for(int k = 0; k < exp_size3; ++k){
                exp_final_set[j * exp_size3 + k] = (*exp_set)[j][k];
            }
        }
    }

    __global__ void get_slice_list(float* encode_ptrs_gpu, int* preorder_ptrs_gpu, int* encode_posis_gpu, int* slice_gpu, int total_len, int* func_flags, int* const_flags){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        int t_n = blockDim.x * gridDim.x;
        int posi_index, idx, total, pre_posi;
        for(int i = tid; i < total_len; i += t_n){
            idx = i;
            pre_posi = encode_posis_gpu[idx];
            posi_index = preorder_ptrs_gpu[i] + pre_posi;
            total = encode_ptrs_gpu[posi_index * 3 + 1];
            if(total > 0){
                func_flags[i] = 1;
                while(total > 0 && idx < total_len){
                    idx += 1;
                    posi_index = preorder_ptrs_gpu[idx] + pre_posi;
                    total += encode_ptrs_gpu[posi_index * 3 + 1] - 1;
                }
            }
            else if(encode_ptrs_gpu[posi_index * 3 + 2] == 2){
                const_flags[i] = 1;
            }
            slice_gpu[i] = idx;
        }
    }

     __global__ void get_expr_list(float* encode_ptrs_gpu, int* preorder_ptrs_gpu, int* encode_posis_gpu, int* slice_gpu, int total_len, int* pre_sums, int* pre_sums_consts, int max_arity, int* exprs_gpu, float* consts_gpu, int init_id_allocator){
        int tid = blockIdx.x * blockDim.x + threadIdx.x;
        int t_n = blockDim.x * gridDim.x;
        int posi_index, idx, total, pre_posi, child_type;
        int unit_len = max_arity + 3;
        int total_exprs = pre_sums[total_len - 1], total_consts = pre_sums_consts[total_len - 1];
        for(int i = total_len - 1; i >= 0; i -= t_n){
            pre_posi = encode_posis_gpu[i];
            posi_index = preorder_ptrs_gpu[i] + pre_posi;
            if(encode_ptrs_gpu[posi_index * 3 + 1] > 0){
                exprs_gpu[(total_exprs - pre_sums[i]) * unit_len] = encode_ptrs_gpu[posi_index * 3];//func
                exprs_gpu[(total_exprs - pre_sums[i]) * unit_len + 1] = encode_ptrs_gpu[posi_index * 3 + 1];//arity
                int offset = i + 1;
                for(int j = 0; j < encode_ptrs_gpu[posi_index * 3 + 1]; ++j){
                    child_type = encode_ptrs_gpu[(preorder_ptrs_gpu[offset] + pre_posi) * 3 + 2];
                    if(child_type == 0){
                        exprs_gpu[(total_exprs - pre_sums[i]) * unit_len + 2 + j] = (total_exprs - pre_sums[offset]) + init_id_allocator;//childs
                    }
                    else if(child_type == 1){
                        exprs_gpu[(total_exprs - pre_sums[i]) * unit_len + 2 + j] = encode_ptrs_gpu[(preorder_ptrs_gpu[offset] + pre_posi) * 3];//childs
                    }
                    else{
                        exprs_gpu[(total_exprs - pre_sums[i]) * unit_len + 2 + j] = -pre_sums_consts[offset];//childs
                        consts_gpu[(total_consts - pre_sums_consts[offset]) - 1] = encode_ptrs_gpu[(preorder_ptrs_gpu[offset] + pre_posi) * 3];
                    }
                    offset = slice_gpu[offset] + 1;
                }
                exprs_gpu[(total_exprs - pre_sums[i]) * unit_len + unit_len - 1] = init_id_allocator + (total_exprs - pre_sums[i]);//arity
            }
        }
    }
}

template<typename scalar_t> 
void TEMPLATE_BIND_FUNCS(py::module& m){
    
    #include <ctime>
    using namespace pygp_utils;
    m.def("test", [](const std::vector<std::vector<py::object>>& res){
        std::vector<std::vector<int>> idxs;
        for(int i = 0; i < res.size(); ++i){
            std::vector<int> idx;
            idx.reserve(res[i].size());
            for(int j = 0; j < res[i].size(); ++j){
                const py::int_& arity = res[i][j].attr("arity"), idx_int = res[i][j].attr("idx");
                if(idx_int != -1){
                    if (arity > 0){
                        idx.push_back(arity + 10);
                    }
                    else{
                        idx.push_back(arity);
                    }
                }
                else{
                    int a;
                }
                // idxs.push_back(res[i][j].attr("arity").cast<int>());
            }
            idxs.push_back(idx);
        }
        printf("here,,,succeed!!!%d\n", res[0][0].attr("arity").cast<int>());
    });
    m.def("tree2graph_v2", [](std::vector<py::array_t<float>> encode_arrays, std::vector<py::array_t<int>> preorder_idxs, std::vector<std::vector<int>> records, std::vector<std::vector<int>> cash_list, int init_id_allocator, int max_arity){
        int* encode_sizes = new int[encode_arrays.size()];
        float* encode_ptrs_gpu;
        int* preorder_ptrs_gpu;
        int* encode_sizes_gpu;
        int* slice_gpu;
        float* consts_gpu;
        int* exprs_gpu;
        // printf("000900\n");
        // hipDeviceSynchronize();
        // hipError_t err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));

        hipStream_t stream_tmp;
        hipStreamCreate(&stream_tmp);

        int total_len = 0;
        for(int i = 0; i < encode_arrays.size(); ++i){
            encode_sizes[i] = total_len;
            total_len += encode_arrays[i].request().shape[0] / 3;
        }
        
        thrust::device_vector<int> func_flags(total_len, 0);
        thrust::device_vector<int> prefix_sum_funcs(total_len);
        
        thrust::device_vector<int> const_flags(total_len, 0);
        thrust::device_vector<int> prefix_sum_consts(total_len);
        // printf("000800\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));

        hipMallocAsync((void**)&slice_gpu, total_len * sizeof(float), stream_tmp); 
        hipMallocAsync((void**)&encode_ptrs_gpu, total_len * 3 * sizeof(float), stream_tmp); 
        hipMallocAsync((void**)&preorder_ptrs_gpu, total_len * sizeof(float), stream_tmp); 
        int* encode_posis = new int[total_len], *encode_posis_gpu;
        int end_posi;
        // printf("000700\n");
        // hipDeviceSynchronize();
        //  err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));

        hipMallocAsync((void**)&encode_posis_gpu, total_len * sizeof(int), stream_tmp); 
        for(int i = 0; i < encode_arrays.size(); ++i){
            hipMemcpyAsync((encode_ptrs_gpu + encode_sizes[i] * 3), encode_arrays[i].request().ptr, encode_arrays[i].request().shape[0] * sizeof(float), hipMemcpyHostToDevice, stream_tmp);
            hipMemcpyAsync((preorder_ptrs_gpu + encode_sizes[i]), preorder_idxs[i].request().ptr, preorder_idxs[i].request().shape[0] * sizeof(int), hipMemcpyHostToDevice, stream_tmp);
            end_posi = (i < (encode_arrays.size() - 1)) ? encode_sizes[i + 1] : total_len;
            for(int j = encode_sizes[i]; j < end_posi; ++j){
                encode_posis[j] = encode_sizes[i];
            }
        }
        hipMemcpyAsync(encode_posis_gpu, encode_posis, total_len * sizeof(int), hipMemcpyHostToDevice, stream_tmp);
        // printf("000700\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));

        int thread_num = total_len < 256 ? int(total_len / 32 + 1) * 32 : 256;
        int batch = ceil(total_len / (thread_num * 10));

        get_slice_list<<<batch, thread_num>>>(encode_ptrs_gpu, preorder_ptrs_gpu, encode_posis_gpu, slice_gpu, total_len, thrust::raw_pointer_cast(func_flags.data()), thrust::raw_pointer_cast(const_flags.data()));
        // printf("00000\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));

        // printf("000100\n");
        thrust::exclusive_scan(func_flags.begin(), func_flags.end(), prefix_sum_funcs.begin());
        thrust::exclusive_scan(const_flags.begin(), const_flags.end(), prefix_sum_consts.begin());
        
        
        // printf("000200\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));

        hipMallocAsync((void**)&consts_gpu, (const_flags[const_flags.size() - 1] + 1) * sizeof(float), stream_tmp); 
        hipMallocAsync((void**)&exprs_gpu, (func_flags[func_flags.size() - 1] + 1) * sizeof(int) * (max_arity + 3), stream_tmp); 
        // printf("000200\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));
        get_expr_list<<<batch, thread_num>>>(encode_ptrs_gpu, preorder_ptrs_gpu, encode_posis_gpu, slice_gpu, total_len, thrust::raw_pointer_cast(prefix_sum_funcs.data()), thrust::raw_pointer_cast(prefix_sum_consts.data()), max_arity, exprs_gpu, consts_gpu, init_id_allocator);
        
        // printf("000300\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));
        py::array_t<int> exprs(func_flags[func_flags.size() - 1] * (max_arity + 3));
        hipMemcpyAsync(exprs.request().ptr, exprs_gpu, func_flags[func_flags.size() - 1] * (max_arity + 3) * sizeof(int), hipMemcpyDeviceToHost, stream_tmp);
        py::array_t<float> consts(const_flags[const_flags.size() - 1]);
        hipMemcpyAsync(consts.request().ptr, consts_gpu, const_flags[const_flags.size() - 1] * sizeof(float), hipMemcpyDeviceToHost, stream_tmp);
        
        delete[] encode_posis;
        delete[] encode_sizes;
        hipStreamDestroy(stream_tmp);

        // printf("000400\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));
        hipFree(slice_gpu);
        hipFree(encode_ptrs_gpu);
        hipFree(preorder_ptrs_gpu);
        hipFree(consts_gpu);
        hipFree(exprs_gpu);
        
        // printf("000500\n");
        // hipDeviceSynchronize();
        // err_l = hipGetLastError();
        // if (err_l != hipSuccess) throw std::runtime_error(hipGetErrorString(err_l));
        return std::tuple<py::array_t<int>, py::array_t<float>>(exprs, consts);
    });
    m.def("tree2graph", [](std::tuple<std::vector<std::string>, std::vector<int>, int> f_attrs, 
                     pybind11::list& ind_after_cashes, pybind11::list& idxs, size_t sym_set_ptr,
                     std::vector<std::vector<int>>& cash_list, std::vector<std::vector<int>>& records, std::vector<int>& paras){
        // printf("idxs: %d\n", idxs.size());
        long max_thread_num = 10;//sysconf(_SC_NPROCESSORS_ONLN) / 10;
        int idxs_len = py::len(idxs);
        if (idxs_len < max_thread_num){
            max_thread_num = 1;
        }
        int ind_num = idxs_len, compute_unit = ceil(float(ind_num) / max_thread_num);
        int batch = ceil(ind_num / compute_unit), cur_posi = 0, cur_ind = 0;
        if (max_thread_num > batch){
            max_thread_num = batch;
        }
        
        std::vector<size_t> idxs_buf(ind_num * 2);
        std::vector<size_t> buf_ind_after_cashes(ind_num * 2);
        py::buffer_info idxs_buf_tmp, buf_ind_after_cashes_tmp;
        for(int k = 0; k < ind_num; ++k){
            idxs_buf_tmp = py::reinterpret_borrow<py::array_t<float>>(idxs[k]).request();
            buf_ind_after_cashes_tmp = py::reinterpret_borrow<py::array_t<int>>(ind_after_cashes[k]).request();
            idxs_buf[k * 2] = size_t(idxs_buf_tmp.ptr), idxs_buf[k * 2 + 1] = size_t(idxs_buf_tmp.shape[0] / 3);
            buf_ind_after_cashes[k * 2] = size_t(buf_ind_after_cashes_tmp.ptr), buf_ind_after_cashes[k * 2 + 1] = size_t(buf_ind_after_cashes_tmp.shape[0]);
        }

        std::thread* t_list = new std::thread[max_thread_num];
        // clock_t st = std::clock();
        // std::vector<std::vector<std::vector<int>>> exp_set_final;
        std::vector<int> record_posi_final;
        std::vector<std::string> record_strs_final;
        std::vector<std::vector<std::vector<int>>> exp_set[batch];
        std::vector<std::string> record_strs[batch];
        std::vector<int> record_posi[batch];
        int id_allocator = paras[1];
        int const_idx = 0;
        std::vector<float> constants;
        for(int k = 0; k < batch - 1; ++k){
            // printf("Batch: %d, %d\n", k, batch);
            // if(k == batch - 1){
            //     compute_unit = ind_num - k * compute_unit;
            // }
            if(t_list[k % max_thread_num].joinable()){
                t_list[k % max_thread_num].join();
            }
            std::tuple<int, int, int> basic_info = std::tuple<int, int, int>(cur_ind, cur_posi, compute_unit);
            t_list[k % max_thread_num] = std::thread(transformer<scalar_t>, &f_attrs, &buf_ind_after_cashes, &idxs_buf, sym_set_ptr, &cash_list, records, &constants, paras, &id_allocator, basic_info, &(exp_set[k]), &(record_posi[k]), &(record_strs[k]), &const_idx);
            // transformer(f_attrs, ind_after_cashes, idxs, sym_set_ptr, cash_list, records, constants, paras, id_allocator, cur_ind, cur_posi, compute_unit, exp_set[k]);
            // if(k < batch - 1){
                for(int i = 0; i < compute_unit; ++i){
                    cur_posi += idxs_buf[(k * compute_unit + i) * 2 + 1];
                }
            // }
            cur_ind += compute_unit;
        }
        compute_unit = ind_num - (batch - 1) * compute_unit;
        std::tuple<int, int, int> basic_info = std::tuple<int, int, int>(cur_ind, cur_posi, compute_unit);
        transformer<scalar_t>(&f_attrs, &buf_ind_after_cashes, &idxs_buf, sym_set_ptr, &cash_list, records, &constants, paras, &id_allocator, basic_info, &(exp_set[batch - 1]), &(record_posi[batch - 1]), &(record_strs[batch - 1]), &const_idx);
        // for(int k = 0; k < max_thread_num; ++k){
        //     if(t_list[k].joinable()){
        //         t_list[k].join();
        //     }
        // }
        // clock_t et = std::clock();
        // printf("t2g time 00 et - st: %f\n", (double)(et - st) / CLOCKS_PER_SEC);
        std::vector<int> layer_info_final;
        layer_info_final.reserve(idxs_buf[1]);
        int exec_len[batch] = {0}, exec_final_len = 0;
        for(int k = 0; k < batch - 1; ++k){
            if(t_list[k % max_thread_num].joinable()){
                t_list[k % max_thread_num].join();
            }
            t_list[k % max_thread_num] = std::thread(exec_sum, &exec_len[k], &exp_set[k]);
        }
        exec_sum(&exec_len[batch - 1], &exp_set[batch - 1]);

        for(int k = 0; k < batch; ++k){
            if(t_list[k % max_thread_num].joinable()){
                t_list[k % max_thread_num].join();
            }
            exec_final_len += exec_len[k];
            for(int z = 0; z < exp_set[k].size(); ++z){
                if(z >= layer_info_final.size()){
                    layer_info_final.push_back(exp_set[k][z].size());
                }
                else{
                    layer_info_final[z] += exp_set[k][z].size();
                }
            }
        }
        exec_final_len *= paras[2];
        py::array_t<int> exp_set_final(exec_final_len);
        size_t exp_set_ptr = size_t(exp_set_final.request().ptr);
        std::vector<size_t> init_posi, accumulate_posi;
        if (layer_info_final.size() > 0){
            init_posi.resize(layer_info_final.size());
            accumulate_posi.resize(layer_info_final.size());
            init_posi[0] = 0;
        }
        for(int i = 1; i < layer_info_final.size(); ++i){
            init_posi[i] = layer_info_final[i - 1] + init_posi[i - 1];
            accumulate_posi[i] = 0;
        }
        for(int k = 0; k < batch; ++k){
            for(int i = 0; i < exp_set[k].size(); ++i){
                exec_cpy(size_t(exp_set_ptr + (init_posi[i] + accumulate_posi[i]) * paras[2] * sizeof(int)), &exp_set[k][i]);
                accumulate_posi[i] += exp_set[k][i].size();
            }
            for(int i = 0; i < record_posi[k].size(); ++i){
                record_posi_final.push_back(record_posi[k][i]);
                record_strs_final.push_back(record_strs[k][i]);
            }
        }
        if(sym_set_ptr == 0){
            delete[] (std::string*)sym_set_ptr;
        }
        
        for(int k = 0; k < max_thread_num; ++k){
            if(t_list[k].joinable()){
                t_list[k].join();
            }
        }
        delete[] t_list;
        // et = std::clock();
        // printf("t2g time et - st: %f\n", (double)(et - st) / CLOCKS_PER_SEC);
    
        return transformer_ret(exp_set_final, layer_info_final, record_posi_final, record_strs_final, constants, id_allocator);
    });
}
PYBIND11_MODULE(pygp_utils, m){
    
    TEMPLATE_BIND_FUNCS<int8_t>(m);
    TEMPLATE_BIND_FUNCS<int32_t>(m);
    TEMPLATE_BIND_FUNCS<int64_t>(m);
    TEMPLATE_BIND_FUNCS<float>(m);
    TEMPLATE_BIND_FUNCS<double>(m);
    // m.def("tree2graph", &transformer);
}